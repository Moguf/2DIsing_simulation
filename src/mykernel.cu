#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

#include "mykernel.hpp"

__global__ void devRandInit(int size,hiprandState *states,long int seed)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = x + y*gridDim.x*blockDim.x;
    if(idx < size){
        hiprand_init(seed,idx,0,&states[idx]);
    }
}