#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

#include "Ising2D.hpp"
#include "mykernel.hpp"

__global__ void devRandInit(int size,hiprandState *states,long int seed)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    long long int idx = x + y*gridDim.x*blockDim.x;
    if( idx < size)
        hiprand_init(seed,idx,0,&states[idx]);
}

__global__ void devSpinInit(int size,hiprandState *states,SPIN *dS)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = x + y*gridDim.x*blockDim.x;
    if(idx < size){
        if(hiprand_uniform(&states[idx]) <= 0.5)
            dS[idx] = -1;
        else
            dS[idx] = 1;
    }
}

__global__ void devCalcEnergy(int J,int *S,int *E,int row,int col)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int nx;
    int ny;

    if( x < row && y < col){
        E[x*col+y] = 0;
        nx = x-1;
        ny = y;
        if(nx < 0)
            nx = col-1;
        E[x*col+y] += -J*S[x*col+y]*S[nx*col+ny];

        nx = x;
        ny = y-1;
        if(ny < 0)
            ny = row-1;
        E[x*col+y] += -J*S[x*col+y]*S[nx*col+ny];

        nx = x+1;
        ny = y;
        if(nx >= row)
            nx = 0;
        E[x*col+y] += -J*S[x*col+y]*S[nx*col+ny];

        nx = x;
        ny = y+1;
        if(ny >= col)
            ny = 0;
        E[x*col+y] += -J*S[x*col+y]*S[nx*col+ny];
     }
    return ;
}