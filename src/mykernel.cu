#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

#include "mykernel.hpp"

__global__ void devRandInit(int size,hiprandState *states,long int seed)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    long long int idx = x + y*gridDim.x*blockDim.x;
    if( idx < size)
        hiprand_init(seed,idx,0,&states[idx]);
}

__global__ void devSpinInit(int size,hiprandState *states,char *dS)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = x + y*gridDim.x*blockDim.x;
    if(idx < size){
        if(hiprand_uniform(&states[idx]) <= 0.5)
            dS[idx] = -1;
        else
            dS[idx] = 1;
    }
}