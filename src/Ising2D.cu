#include <iostream>
#include <array>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "Ising2D.hpp"
#include "mykernel.hpp"

void Ising2D::devInfo(){
    int dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,dev);

    printf("Device %d: %s\n",dev,deviceProp.name);
    

}

void Ising2D::hostInit(){
    hS = (char *)malloc(sizeof(char)*size);
    hE = (char *)malloc(sizeof(char)*size);
}

void Ising2D::devInit(){
    hipMalloc((char **)&dS,sizeof(char)*size);
    hipMalloc((char **)&dE,sizeof(char)*size);
}

void Ising2D::hostEnd(){
    free(hS);
    free(hE);
}

void Ising2D::devEnd(){
    hipFree(dS);
    hipFree(dE);
}

void Ising2D::run(){

}


