#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

#include "mykernel.hpp"
#include "Ising2D.hpp"

using namespace std;
#define D_CHECK(err){\
    if(err !=hipSuccess)\
        printf("error %s,at %d\n",hipGetErrorString(err),__LINE__);    \
}

#define ROW 3072
#define COL 3072

struct Environments{
    int i;
}typedef Env;

class Ising2D{
private:
    Env env;

public:
    inline void run();
    inline void devInit();
    inline void hostInit();
    inline void devEnd();
    inline void hostEnd();
};

int main(void){

    return 0;
}
    
