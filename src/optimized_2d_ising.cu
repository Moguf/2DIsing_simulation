#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

#include "Ising2D.hpp"
#include "mykernel.hpp"

using namespace std;

int main(void){
    Ising2D tmp;
    tmp.hostInit();
    tmp.devInit();
    tmp.run();
    tmp.devEnd();
    tmp.hostEnd();

    return 0;
}
    
